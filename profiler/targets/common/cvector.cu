/*
 *  Copyright (c) 2018-2020, Carnegie Mellon University
 *  See LICENSE for details
 */
/***************************************************************************
 * SPL Matrix                                                              *
 *                                                                         *
 * Computes matrix that corresponds to SPL generated routine               *
 ***************************************************************************/

#include <limits.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include <hip/hip_runtime_api.h>

#ifndef MIN
#define MIN(a,b) (((a)<(b))?(a):(b))
#endif

#ifndef ROWS
#error ROWS must be defined
#endif
#ifndef COLUMNS
#error COLUMNS must be defined
#endif

hipfftDoubleReal  *Input, *Output;
hipfftDoubleReal  *dev_in, *dev_out;


void initialize(int argc, char **argv) {

	// In many case ROWS & COLUMNS are equal; however, when they are not it is
	// important to use the correct one when allocating memory for the in/out
	// buffers.  The *input* buffer should be dimensioned by COLUMNS, while the
	// *output* buffer should be dimensioned by ROWS
	
	hipHostMalloc ( &Input,  sizeof(hipfftDoubleReal) * COLUMNS );
	checkCudaErrors(hipGetLastError());
	hipHostMalloc ( &Output, sizeof(hipfftDoubleReal) * ROWS );
	checkCudaErrors(hipGetLastError());

	hipMalloc     ( &dev_in,  sizeof(hipfftDoubleReal) * COLUMNS );
	checkCudaErrors(hipGetLastError());
	hipMalloc     ( &dev_out, sizeof(hipfftDoubleReal) * ROWS );
	checkCudaErrors(hipGetLastError());

	INITFUNC();
}

void finalize() {
	hipHostFree (Output);
	hipHostFree (Input);
	hipFree     (dev_out);
	hipFree     (dev_in);
}

void compute_vector()
{
	int indx;
	printf("[ ");

	hipMemcpy ( dev_in, Input, sizeof(hipfftDoubleReal) * COLUMNS, hipMemcpyHostToDevice);
	checkCudaErrors(hipGetLastError());

	FUNC(dev_out, dev_in);
	checkCudaErrors(hipGetLastError());
	hipDeviceSynchronize();

	hipMemcpy ( Output, dev_out, sizeof(hipfftDoubleReal) * ROWS, hipMemcpyDeviceToHost);
	checkCudaErrors(hipGetLastError());

	for (indx = 0; indx < ROWS; indx++) {
		if (indx != 0) {
			printf(", ");
		}
		printf("FloatString(\"%.18g\")", Output[indx]);
	}
	printf("];\n");
}



int main(int argc, char** argv) {

	initialize(argc, argv);

	int tlen = sizeof(testvector) / sizeof(testvector[0]);
	
	for (int i = 0; i < MIN(tlen, COLUMNS); i++) {
		Input[i] = (hipfftDoubleReal)testvector[i];
	}
	
	compute_vector();
	finalize();
	return EXIT_SUCCESS;
}
