/*
 *  Copyright (c) 2018-2020, Carnegie Mellon University
 *  See LICENSE for details
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>

#ifdef WIN64
#include <winsock.h>			// defines struct timeval -- go figure?
#include <sys/timeb.h>
#endif							// WIN64

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include <hip/hip_runtime_api.h>

#ifndef DESTROYFUNC
#define DESTROYFUNC destroy_sub
#endif

// ===  Test SPIRAL ===

void setup_spiral_test()
{
    checkCudaErrors(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
    // printf("Running SPIRAL Hello World CUDA example...\n");

    INITFUNC();
}

void teardown_spiral_test()
{
    DESTROYFUNC();
}

void test_spiral(double* in, double* out)
{

    FUNC(out, in);
    checkCudaErrors(hipGetLastError());
}


int main(int argc, char** argv)
{
	hipfftDoubleReal  *out, *in;
	hipfftDoubleReal  *dev_out, *dev_in;
	hipEvent_t      begin, end;

#ifdef WIN64
    struct timeb     start, finish;
    ftime(&start);
#else
	struct timespec  start, finish;
	clock_gettime(CLOCK_MONOTONIC, &start);
#endif					 // WIN64

	// In many case ROWS & COLUMNS are equal; however, when they are not it is
	// important to use the correct one when allocating memory for the in/out
	// buffers.  The *input* buffer should be dimensioned by COLUMNS, while the
	// *output* buffer should be dimensioned by ROWS
	
	hipEventCreate ( &begin );
	hipEventCreate ( &end );
	hipHostMalloc  ( &in,      sizeof(hipfftDoubleReal) * COLUMNS );
	hipHostMalloc  ( &out,     sizeof(hipfftDoubleReal) * ROWS );
	hipMalloc      ( &dev_in,  sizeof(hipfftDoubleReal) * COLUMNS );
	hipMalloc      ( &dev_out, sizeof(hipfftDoubleReal) * ROWS );

	for (int i = 0; i < /* ROWS */ COLUMNS; i++)
		in[i] = i;

	setup_spiral_test();
	hipMemcpy ( dev_in, in,   sizeof(hipfftDoubleReal) * COLUMNS, hipMemcpyHostToDevice);
	checkCudaErrors(hipGetLastError());
	
	checkCudaErrors( hipEventRecord(begin) );
	test_spiral(dev_in, dev_out);
	checkCudaErrors( hipEventRecord(end) );
	hipDeviceSynchronize();
	hipMemcpy ( out, dev_out, sizeof(hipfftDoubleReal) * ROWS, hipMemcpyDeviceToHost);
	checkCudaErrors(hipGetLastError());

	teardown_spiral_test();

	float milli = 0.0;
	checkCudaErrors ( hipEventElapsedTime ( &milli, begin, end ) );
	printf("%f;\t\t##  CuFFT-based execution [ms]\n", milli);

#ifdef WIN64
    ftime(&finish);
    double elapsed = (1000.0 * (finish.time - start.time)) + (finish.millitm - start.millitm);
	printf("%f;\t\t##  elapsed time [ms]\n", elapsed);
#else
	// time for non-Windows systems
	clock_gettime(CLOCK_MONOTONIC, &finish);
	double elapsed = ( ( (double)finish.tv_sec * 1e9 + (double)finish.tv_nsec) -
					   ( (double)start.tv_sec  * 1e9 + (double)start.tv_nsec ) );
	printf("%f;\t\t##  elapsed time [ms]\n", elapsed * 1e-6 );
#endif // WIN64

	fflush(stdout);
	
	hipHostFree  ( in );
	hipHostFree  ( out );
	hipFree      ( dev_in );
	hipFree      ( dev_out );
	
	return EXIT_SUCCESS;
}
						 
