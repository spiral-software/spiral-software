#include <hip/hip_runtime_api.h>

int main ( int argc, char* argv[] )
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	printLastCudaError("looking for CUDA devices: "); //hipGetLastError()

	if (deviceCount == 0) {
		printf("No GPU devices found, exit abnormally\n");
		return (-1);
	}
	
	int device;
	for (device = 0; device < deviceCount; device++) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, device);
		printf("Device %d has compute capability %d.%d.\n",
			   device, deviceProp.major, deviceProp.minor);
	}
	return 0;
}
